
#include <hip/hip_runtime.h>
#ifdef GALAX_MODEL_GPU

#include "cuda.h"
#include "kernel.cuh"
#define DIFF_T (0.1f)
#define EPS (1.0f)

inline __host__ __device__ float3 operator-(float3 a, float3 b)
{
    return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}
inline __host__ __device__ float3 operator*(float3 a, float b)
{
    return make_float3(a.x * b, a.y * b, a.z * b);
}
inline __host__ __device__ void operator+=(float3 &a, float3 b)
{
    a.x += b.x;
    a.y += b.y;
    a.z += b.z;
}
__global__ void compute_acc(float3 * positionsGPU, float3 * velocitiesGPU, float3 * accelerationsGPU, float* massesGPU, int n_particles)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	// unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	accelerationsGPU[i].x = 0.0f;
	accelerationsGPU[i].y = 0.0f;
	accelerationsGPU[i].z = 0.0f;
	for (int j = 0; j < n_particles; j++)
	{
		if(i != j)
		{
			const float diffx = positionsGPU[j].x - positionsGPU[i].x;
			const float diffy = positionsGPU[j].y - positionsGPU[i].y;
			const float diffz = positionsGPU[j].z - positionsGPU[i].z;

			float dij = diffx * diffx + diffy * diffy + diffz * diffz;

			if (dij < 1.0)
			{
				dij = 10.0;
			}
			else
			{
				dij = sqrt(dij);
				dij = 10.0 / (dij * dij * dij);
			}

			accelerationsGPU[i].x += diffx * dij * massesGPU[j];
			accelerationsGPU[i].y += diffy * dij * massesGPU[j];
			accelerationsGPU[i].z += diffz * dij * massesGPU[j];
		}
	}
	
	
}

__global__ void maj_pos(float3 * positionsGPU, float3 * velocitiesGPU, float3 * accelerationsGPU)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	velocitiesGPU[i].x += accelerationsGPU[i].x * 2.0f;
	velocitiesGPU[i].y += accelerationsGPU[i].y * 2.0f;
	velocitiesGPU[i].z += accelerationsGPU[i].z * 2.0f;
	positionsGPU[i].x += velocitiesGPU   [i].x * DIFF_T;
	positionsGPU[i].y += velocitiesGPU   [i].y * DIFF_T;
	positionsGPU[i].z += velocitiesGPU   [i].z * DIFF_T;

}

void update_position_cu(float3* positionsGPU, float3* velocitiesGPU, float3* accelerationsGPU, float* massesGPU, int n_particles)
{
	int nthreads = 128;
	int nblocks =  (n_particles + (nthreads -1)) / nthreads;

	compute_acc<<<nblocks, nthreads>>>(positionsGPU, velocitiesGPU, accelerationsGPU, massesGPU, n_particles);
	maj_pos    <<<nblocks, nthreads>>>(positionsGPU, velocitiesGPU, accelerationsGPU);
}

#endif // GALAX_MODEL_GPU